#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <NvInfer.h>
#include <opencv2/opencv.hpp>
#include <chrono>
#include <ros/ros.h>
#include <utils.h>

using namespace nvinfer1;

class Logger : public ILogger {
  void log(Severity severity, const char * msg) override {
    if (severity != Severity::kINFO)
      ROS_INFO("%s", msg);
  }
} gLogger;

IRuntime *runtime;
ICudaEngine *engine;
IExecutionContext *context;
int inputBindingIndex, outputBindingIndex;
int inputHeight, inputWidth;
Dims inputDims, outputDims;
bool is_initialized = false;
void *bindings[2];

// pointers
size_t numInput, numOutput;
float *inputDataHost, *outputDataHost;
float *inputDataDevice, *outputDataDevice;

/* void setup(std::string planFilename, std::string inputName, std::string outputName) { */
void setup() {
  ROS_INFO("setup");
  std::ifstream planFile("/home/ubuntu/tensorrt/resnet_test/resnet_v1_50_finetuned_4class_altered_model.plan");
  if(!planFile.is_open()) {
    ROS_INFO("cannot get plan file");
    is_initialized = false;
  } else {
    std::stringstream planBuffer;
    planBuffer << planFile.rdbuf();
    std::string plan = planBuffer.str();

    runtime = createInferRuntime(gLogger);
    engine  = runtime->deserializeCudaEngine((void*)plan.data(), plan.size(), nullptr);
    context = engine->createExecutionContext();
    ROS_INFO("load setup finished");

    inputBindingIndex = engine->getBindingIndex("images");
    outputBindingIndex = engine->getBindingIndex("resnet_v1_50/SpatialSqueeze");
    inputDims = engine->getBindingDimensions(inputBindingIndex);
    outputDims = engine->getBindingDimensions(outputBindingIndex);
    inputHeight = inputDims.d[1];
    inputWidth = inputDims.d[2];
    ROS_INFO("input: %d, %d", inputHeight, inputWidth);

    numInput = numTensorElements(inputDims);
    numOutput = numTensorElements(outputDims);

    // host
    inputDataHost = (float*) malloc(numInput * sizeof(float));
    outputDataHost = (float*) malloc(numOutput * sizeof(float));
    // device
    hipMalloc(&inputDataDevice, numInput * sizeof(float));
    hipMalloc(&outputDataDevice, numOutput * sizeof(float));

    is_initialized = true;
    ROS_INFO("initialize finished %d, %d", numInput, numOutput);
  }
}

void destroy(void) {
  if(is_initialized) {
    runtime->destroy();
    engine->destroy();
    context->destroy();
    free(inputDataHost);
    free(outputDataHost);
    hipFree(inputDataDevice);
    hipFree(outputDataDevice);
  }
  is_initialized = false;
}

void infer(cv::Mat image, float* out) {
  // cvの画像からcnnを走らせる
  ROS_INFO("get");
  cv::resize(image, image, cv::Size(inputWidth, inputHeight));
  cvImageToTensor(image, inputDataHost, inputDims);
  preprocessVgg(inputDataHost, inputDims);
  bindings[inputBindingIndex] = (void*)inputDataDevice;
  bindings[outputBindingIndex] = (void*)outputDataDevice;

  hipMemcpy(inputDataDevice, inputDataHost, numInput * sizeof(float), hipMemcpyHostToDevice);
  context->execute(1, bindings);
  hipMemcpy(outputDataHost, outputDataDevice, numOutput * sizeof(float), hipMemcpyDeviceToHost);
  // output
  ROS_INFO("%f %f %f %f", outputDataHost[0], outputDataHost[1], outputDataHost[2], outputDataHost[3]);
  for (int i = 0; i < 4; i++) {
    out[i] = outputDataHost[i];
  }
}

void test(void) {
  ROS_INFO("inside cu");
  hipDeviceSynchronize();
}



